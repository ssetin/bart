#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

/*!
    vectors for processing in gpu memory
*/
namespace cuda_data{
    //double *dev_w=NULL;
    double *dev_roww=NULL;
    double *dev_x=NULL;
}

/*!
    CUDA Kernel Device code
    for(int col=0;col<sizex;col++){
        w[row][col]=w[row][col] + d*x[col];
    }
    w[row*sizey+i]+=d*x[i];
    w[idx]+=d*x[i];
*/
__global__ void vectorCorrect(double *w, const double *x, const int sizex, const int sizey, const double d, const int row){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if(idx<sizex){
        w[idx]+=d*x[idx];
    }

}

/*!
    Free memory
*/
extern "C" bool freeobjects_cuda(){
    if(cuda_data::dev_x!=NULL)
        hipFree(cuda_data::dev_x);
    if(cuda_data::dev_roww!=NULL)
        hipFree(cuda_data::dev_roww);
    return true;
}

/*!
    Allocate cuda_data
*/
extern "C" bool allocateobjects_cuda(const int sizex, const int sizey, double** w){

    hipError_t err = hipSuccess;

    err = hipMalloc((void**)&cuda_data::dev_roww, sizex*sizeof(double));
    if (err != hipSuccess){
        fprintf(stderr, "allocateobjects_cuda/hipMalloc dev_w returned error %s (code %d), line(%d)\n", hipGetErrorString(err), err, __LINE__);
        freeobjects_cuda();
        return false;
    }

    err = hipMalloc((void**)&cuda_data::dev_x, sizex * sizeof(double));
    if (err != hipSuccess){
        fprintf(stderr, "allocateobjects_cuda/hipMalloc dev_x returned error %s (code %d), line(%d)\n", hipGetErrorString(err), err, __LINE__);
        freeobjects_cuda();
        return false;
    }

    return true;
}



/*!
    Prepare data to calculate on GPU
*/
extern "C" bool correctweight_cuda(double **w,const double *x,const int sizex, const int sizey,const int row,const double d){

    hipError_t err = hipSuccess;
    size_t size = sizex * sizeof(double);        

    err=hipMemcpy(cuda_data::dev_roww, w[row], size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "correctweight_cuda/hipMemcpy dev_roww returned error %s (code %d), line(%d)\n", hipGetErrorString(err), err, __LINE__);
        freeobjects_cuda();
        return false;
    }

    err=hipMemcpy(cuda_data::dev_x, x, size, hipMemcpyHostToDevice);    
    if (err != hipSuccess){
        fprintf(stderr, "correctweight_cuda/hipMemcpy dev_x returned error %s (code %d), line(%d)\n", hipGetErrorString(err), err, __LINE__);
        freeobjects_cuda();
        return false;
    }

    int threadsPerBlock = 256;
    int blocksPerGrid =(sizex + threadsPerBlock - 1) / threadsPerBlock;

    vectorCorrect<<<blocksPerGrid, threadsPerBlock>>>(cuda_data::dev_roww, cuda_data::dev_x, sizex, sizey, d, row);

    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch vectorCorrect kernel (error code %s)!\n", hipGetErrorString(err));
        freeobjects_cuda();
        return false;
    }    

    //err=hipMemcpy(w[row], &cuda_data::dev_w[row*sizey], size, hipMemcpyDeviceToHost);
    err=hipMemcpy(w[row], cuda_data::dev_roww, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess){
        fprintf(stderr, "correctweight_cuda/hipMemcpy dev_roww returned error %s (code %d), line(%d)\n", hipGetErrorString(err), err, __LINE__);
        freeobjects_cuda();
        return false;
    }

    return true;

}
