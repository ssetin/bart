#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


const int BLOCK_SIZE=256;

/*!
    vectors for processing in gpu memory
*/
namespace cuda_data{
    double *dev_w=NULL;
    double *dev_x=NULL;
}

/*!
    CUDA Kernel Device code
    for(int col=0;col<sizex;col++){
        w[row][col]=w[row][col] + d*x[col];
    }
    w[row*sizey+idx]+=d*x[idx];
*/
__global__ void vectorCorrect(double *w, const double *x, const int sizex, const int sizey,const double d, const int row){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ double shared_w[BLOCK_SIZE];
    __shared__ double shared_x[BLOCK_SIZE];

    if(idx<sizex){

        shared_x[threadIdx.x]=x[idx];
        shared_w[threadIdx.x]=w[row*sizey+idx];

        __syncthreads();

        shared_w[threadIdx.x]+=d*shared_x[threadIdx.x];

        __syncthreads();

        w[row*sizey+idx]=shared_w[threadIdx.x];
    }


}

/*!
    Free memory
*/
extern "C" bool freeobjects_cuda(){
    if(cuda_data::dev_x!=NULL)
        hipFree(cuda_data::dev_x);
    if(cuda_data::dev_w!=NULL)
        hipFree(cuda_data::dev_w);
    return true;
}

/*!
    Allocate cuda_data
*/
extern "C" bool allocateobjects_cuda(const int sizex, const int sizey, double *w){
    hipError_t err = hipSuccess;

    err = hipMalloc((void**)&cuda_data::dev_w,  sizex*sizey*sizeof(double));
    if (err != hipSuccess){
        fprintf(stderr, "allocateobjects_cuda/hipMalloc dev_w returned error %s (code %d), line(%d)\n", hipGetErrorString(err), err, __LINE__);
        freeobjects_cuda();
        return false;
    }

    err = hipMalloc((void**)&cuda_data::dev_x, sizex * sizeof(double));
    if (err != hipSuccess){
        fprintf(stderr, "allocateobjects_cuda/hipMalloc dev_x returned error %s (code %d), line(%d)\n", hipGetErrorString(err), err, __LINE__);
        freeobjects_cuda();
        return false;
    }

    err=hipMemcpy(cuda_data::dev_w, w, sizex *sizey * sizeof(double) , hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "allocateobjects_cuda/hipMemcpy dev_w returned error %s (code %d), line(%d)\n", hipGetErrorString(err), err, __LINE__);
        freeobjects_cuda();
        return false;
    }

    return true;
}


extern "C" bool setx_cuda(const int sizex, double *x){
    hipError_t err = hipSuccess;
    err=hipMemcpy(cuda_data::dev_x, x, sizex * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "setx_cuda/hipMemcpy dev_x returned error %s (code %d), line(%d)\n", hipGetErrorString(err), err, __LINE__);
        freeobjects_cuda();
        return false;
    }
    return true;
}



/*!
    Prepare data to calculate on GPU
*/
extern "C" bool correctweight_cuda(double *w,const int sizex, const int sizey,const int row,const double d){

    hipError_t err = hipSuccess;
    size_t size = sizex * sizeof(double);    

    dim3 threadsPerBlock(BLOCK_SIZE, 1);
    dim3 blocksPerGrid(sizex / BLOCK_SIZE +1, 1);

    vectorCorrect<<<blocksPerGrid, threadsPerBlock>>>(cuda_data::dev_w, cuda_data::dev_x, sizex, sizey, d, row);

    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch vectorCorrect kernel (error code %s)!\n", hipGetErrorString(err));
        freeobjects_cuda();
        return false;
    }    

    hipDeviceSynchronize();


    err=hipMemcpyAsync(&w[row*sizey], &cuda_data::dev_w[row*sizey], size, hipMemcpyDeviceToHost);

    if (err != hipSuccess){
        fprintf(stderr, "correctweight_cuda/hipMemcpy dev_w returned error %s (code %d), line(%d)\n", hipGetErrorString(err), err, __LINE__);
        freeobjects_cuda();
        return false;
    }


    return true;

}
